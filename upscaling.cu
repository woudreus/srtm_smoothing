#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <>
#include <arrayfire.h>
#include <af/hip/hip_runtime.h>

extern "C"
__declspec(dllexport)
void bicubUpscaling(double *x,	double *outx, int *ncol, int *n);

void bicubUpscaling(double *x, double *outx, int *ncol, int *n) {
	// initialize device memory variables
	double *d_x;

	//input memory allocation
	hipMalloc((void**)&d_x, (*n / 9) * sizeof(double));

	//copy host memory to allocated device memory
	hipMemcpy(d_x, x, (*n / 9) * sizeof(double), hipMemcpyHostToDevice);

	//conduct AF operations
	af::array d_A((*ncol / 3), (*ncol / 3), d_x, afDevice);
	af::eval(d_A);
	af::sync();
	d_A = resize(3, d_A, AF_INTERP_BICUBIC);

	//return AF arrays to device memory 
	double *x_interp = d_A.device<double>();

	//copy device memory to host memory
	hipMemcpy(outx, x_interp, *n * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_x);
	hipFree(x_interp);

}
